// SPDX-FileCopyrightText: Copyright (c) 2024-2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
// SPDX-License-Identifier: Apache-2.0
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#include <cstring>
#include <memory>
#include <vector>

// Error checking macro
#define CUDA_CHECK(call)                                                                            \
  do {                                                                                              \
    hipError_t error = call;                                                                       \
    if (error != hipSuccess) {                                                                     \
      fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
      return error;                                                                                 \
    }                                                                                               \
  } while (0)

// Number of elements to process per thread
#define ELEMENTS_PER_THREAD 4

// Use cache-line sized chunks when possible
#define CACHE_LINE_SIZE 128  // 128 bytes for most GPUs

// Optimized kernel that processes elements in a dimension-aware manner
__global__ void
copy_blocks_kernel(
    const void* src_data, void* dst_data, const int* src_block_ids, const int* dst_block_ids, int num_block_pairs,
    int prefix_dim, int suffix_dim, int elem_size, size_t src_prefix_stride, size_t src_block_stride,
    size_t src_suffix_stride, size_t dst_prefix_stride, size_t dst_block_stride, size_t dst_suffix_stride)
{
  // Calculate the total number of elements to process
  const size_t total_elements = (size_t)prefix_dim * num_block_pairs * suffix_dim;

  // Calculate the total number of bytes in the suffix part
  const size_t bytes_per_suffix = (size_t)suffix_dim * elem_size;

  // Calculate how many cache-line sized chunks per suffix part
  const size_t chunks_per_suffix = (bytes_per_suffix + CACHE_LINE_SIZE - 1) / CACHE_LINE_SIZE;
  const size_t elements_per_chunk = CACHE_LINE_SIZE / elem_size;
  const bool is_perfect_chunk = (bytes_per_suffix % CACHE_LINE_SIZE) == 0;

  // Get global thread index
  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Each thread processes ELEMENTS_PER_THREAD chunk indices
  const size_t start_chunk = thread_idx * ELEMENTS_PER_THREAD;
  const size_t total_chunks = prefix_dim * num_block_pairs * chunks_per_suffix;

  // Early exit if completely out of range
  if (start_chunk >= total_chunks) {
    return;
  }

  // Process multiple chunks per thread
  for (int chunk_offset = 0; chunk_offset < ELEMENTS_PER_THREAD; chunk_offset++) {
    // Current chunk index
    size_t chunk_idx = start_chunk + chunk_offset;

    // Check if this chunk is within bounds
    if (chunk_idx >= total_chunks) {
      return;  // No more chunks to process
    }

    // Decompose chunk index into prefix, block, and suffix chunks
    size_t blocks_chunks = num_block_pairs * chunks_per_suffix;
    size_t prefix_idx = chunk_idx / blocks_chunks;
    size_t remainder = chunk_idx % blocks_chunks;
    size_t block_pair_idx = remainder / chunks_per_suffix;
    size_t chunk_in_suffix = remainder % chunks_per_suffix;

    // Bounds check
    if (prefix_idx >= prefix_dim || block_pair_idx >= num_block_pairs) {
      continue;  // Skip this chunk
    }

    // Get the actual source and destination block IDs
    int src_block_id = src_block_ids[block_pair_idx];
    int dst_block_id = dst_block_ids[block_pair_idx];

    // Calculate element offset within the suffix dimension
    size_t suffix_elem_offset = chunk_in_suffix * CACHE_LINE_SIZE / elem_size;

    // Calculate the byte offset using explicit strides for each dimension
    size_t src_byte_offset =
        prefix_idx * src_prefix_stride + src_block_id * src_block_stride + suffix_elem_offset * src_suffix_stride;

    size_t dst_byte_offset =
        prefix_idx * dst_prefix_stride + dst_block_id * dst_block_stride + suffix_elem_offset * dst_suffix_stride;

    // Calculate elements to copy in this chunk
    size_t elements_to_copy = elements_per_chunk;
    if (!is_perfect_chunk && chunk_in_suffix == chunks_per_suffix - 1) {
      // Last chunk might be smaller
      elements_to_copy = suffix_dim - suffix_elem_offset;
    }

    // Copy data based on element size for better performance
    if (elem_size == 2 && (elements_to_copy % 2 == 0)) {
      // Use 32-bit loads/stores for 16-bit data when possible (half precision)
      const uint32_t* src_ptr = (const uint32_t*)((const char*)src_data + src_byte_offset);
      uint32_t* dst_ptr = (uint32_t*)((char*)dst_data + dst_byte_offset);

      for (size_t i = 0; i < elements_to_copy / 2; i++) {
        dst_ptr[i] = src_ptr[i];
      }
      // } else if (elem_size == 1 && (elements_to_copy % 4 == 0)) {
      //   // Use 32-bit loads/stores for 8-bit data when possible (half precision)
      //   const uint32_t* src_ptr = (const uint32_t*)((const char*)src_data + src_byte_offset);
      //   uint32_t* dst_ptr = (uint32_t*)((char*)dst_data + dst_byte_offset);

      //   for (size_t i = 0; i < elements_to_copy / 4; i++) {
      //     dst_ptr[i] = src_ptr[i];
      //   }
    } else if (elem_size == 2) {
      // Handle 16-bit elements one by one if necessary
      const uint16_t* src_ptr = (const uint16_t*)((const char*)src_data + src_byte_offset);
      uint16_t* dst_ptr = (uint16_t*)((char*)dst_data + dst_byte_offset);

      for (size_t i = 0; i < elements_to_copy; i++) {
        dst_ptr[i] = src_ptr[i];
      }
    } else if (elem_size == 4) {
      // Copy 32-bit elements (float, int32)
      const uint32_t* src_ptr = (const uint32_t*)((const char*)src_data + src_byte_offset);
      uint32_t* dst_ptr = (uint32_t*)((char*)dst_data + dst_byte_offset);

      for (size_t i = 0; i < elements_to_copy; i++) {
        dst_ptr[i] = src_ptr[i];
      }
    } else if (elem_size == 8) {
      // Copy 64-bit elements (double, int64)
      const uint64_t* src_ptr = (const uint64_t*)((const char*)src_data + src_byte_offset);
      uint64_t* dst_ptr = (uint64_t*)((char*)dst_data + dst_byte_offset);

      for (size_t i = 0; i < elements_to_copy; i++) {
        dst_ptr[i] = src_ptr[i];
      }
    } else {
      // For other element sizes, copy byte by byte
      const char* src_ptr = (const char*)src_data + src_byte_offset;
      char* dst_ptr = (char*)dst_data + dst_byte_offset;

      for (size_t i = 0; i < elements_to_copy * elem_size; i++) {
        dst_ptr[i] = src_ptr[i];
      }
    }
  }
}

// Simplified launcher that uses the 3D tensor view
extern "C" hipError_t
copy_blocks_launcher_3d(
    const void* src_data, void* dst_data, const int* d_src_block_ids, const int* d_dst_block_ids, int num_block_pairs,
    int prefix_dim, int suffix_dim, int elem_size, int src_block_dim, int dst_block_dim, hipStream_t stream)
{
  // Validate inputs
  if (src_data == NULL || dst_data == NULL) {
    fprintf(stderr, "NULL data pointers\n");
    return hipErrorInvalidValue;
  }

  if (d_src_block_ids == NULL || d_dst_block_ids == NULL) {
    fprintf(stderr, "NULL device block ID pointers\n");
    return hipErrorInvalidValue;
  }

  if (num_block_pairs <= 0) {
    fprintf(stderr, "Invalid number of block pairs: %d\n", num_block_pairs);
    return hipErrorInvalidValue;
  }

  if (prefix_dim <= 0 || suffix_dim <= 0 || elem_size <= 0) {
    fprintf(stderr, "Invalid dimensions: prefix=%d, suffix=%d, elem=%d\n", prefix_dim, suffix_dim, elem_size);
    return hipErrorInvalidValue;
  }

  // Calculate row-major strides internally
  size_t src_suffix_stride = elem_size;
  size_t dst_suffix_stride = elem_size;

  size_t src_block_stride = suffix_dim * src_suffix_stride;
  size_t dst_block_stride = suffix_dim * dst_suffix_stride;

  size_t src_prefix_stride = src_block_dim * src_block_stride;
  size_t dst_prefix_stride = dst_block_dim * dst_block_stride;

  // // Optional debug output
  // printf(
  //     "Tensor dims: prefix=%d, src_blocks=%d, dst_blocks=%d, suffix=%d, elem_size=%d\n", prefix_dim, src_blocks_dim,
  //     dst_blocks_dim, suffix_dim, elem_size);
  // printf(
  //     "Calculated strides: src_prefix=%zu, src_block=%zu, src_suffix=%zu\n", src_prefix_stride, src_block_stride,
  //     src_suffix_stride);

  // Calculate total number of bytes to copy
  size_t total_bytes = (size_t)prefix_dim * num_block_pairs * suffix_dim * elem_size;

  // Calculate number of cache-line sized chunks
  size_t bytes_per_suffix = (size_t)suffix_dim * elem_size;
  size_t chunks_per_suffix = (bytes_per_suffix + CACHE_LINE_SIZE - 1) / CACHE_LINE_SIZE;
  size_t total_chunks = prefix_dim * num_block_pairs * chunks_per_suffix;

  // Adjust grid size to account for multiple elements per thread
  int total_threads = (total_chunks + ELEMENTS_PER_THREAD - 1) / ELEMENTS_PER_THREAD;
  int cuda_block_size = 256;
  int grid_size = (total_threads + cuda_block_size - 1) / cuda_block_size;

  // Validate grid size
  if (grid_size <= 0) {
    fprintf(stderr, "Invalid grid size: %d\n", grid_size);
    return hipErrorInvalidValue;
  }

  // Launch kernel on specified stream
  copy_blocks_kernel<<<grid_size, cuda_block_size, 0, stream>>>(
      src_data, dst_data, d_src_block_ids, d_dst_block_ids, num_block_pairs, prefix_dim, suffix_dim, elem_size,
      src_prefix_stride, src_block_stride, src_suffix_stride, dst_prefix_stride, dst_block_stride, dst_suffix_stride);

  // Check for kernel launch errors immediately
  hipError_t kernel_error = hipGetLastError();
  if (kernel_error != hipSuccess) {
    fprintf(stderr, "Kernel execution error: %s\n", hipGetErrorString(kernel_error));
    return kernel_error;
  }

  return hipSuccess;
}


extern "C" hipError_t
copy_blocks_memcpy_3d(
    const void* src_data, void* dst_data, const int* h_src_block_ids, const int* h_dst_block_ids, int num_block_pairs,
    int prefix_dim, int suffix_dim, int elem_size, int src_block_dim, int dst_block_dim, hipStream_t stream)
{
  // Validate inputs
  if (src_data == NULL || dst_data == NULL) {
    fprintf(stderr, "NULL data pointers\n");
    return hipErrorInvalidValue;
  }

  if (h_src_block_ids == NULL || h_dst_block_ids == NULL) {
    fprintf(stderr, "NULL host block ID pointers\n");
    return hipErrorInvalidValue;
  }

  if (num_block_pairs <= 0) {
    fprintf(stderr, "Invalid number of block pairs: %d\n", num_block_pairs);
    return hipErrorInvalidValue;
  }

  if (prefix_dim <= 0 || suffix_dim <= 0 || elem_size <= 0) {
    fprintf(stderr, "Invalid dimensions: prefix=%d, suffix=%d, elem=%d\n", prefix_dim, suffix_dim, elem_size);
    return hipErrorInvalidValue;
  }

  // Calculate row-major strides for source and destination
  size_t suffix_size_bytes = suffix_dim * elem_size;
  size_t src_block_stride = suffix_size_bytes;
  size_t dst_block_stride = suffix_size_bytes;
  size_t src_prefix_stride = src_block_dim * src_block_stride;
  size_t dst_prefix_stride = dst_block_dim * dst_block_stride;

  size_t count = 0;

  // Loop through all prefix dimensions and block pairs
  for (int prefix_idx = 0; prefix_idx < prefix_dim; prefix_idx++) {
    for (int pair_idx = 0; pair_idx < num_block_pairs; pair_idx++) {
      int src_block_id = h_src_block_ids[pair_idx];
      int dst_block_id = h_dst_block_ids[pair_idx];

      // Calculate byte offsets
      size_t src_offset = prefix_idx * src_prefix_stride + src_block_id * src_block_stride;
      size_t dst_offset = prefix_idx * dst_prefix_stride + dst_block_id * dst_block_stride;

      // Copy the suffix data in one call (it's contiguous)
      const void* src_ptr = static_cast<const char*>(src_data) + src_offset;
      void* dst_ptr = static_cast<char*>(dst_data) + dst_offset;

      hipError_t error = hipMemcpyAsync(dst_ptr, src_ptr, suffix_size_bytes, hipMemcpyDefault, stream);
      if (error != hipSuccess) {
        return error;
      }

      count += suffix_size_bytes;
    }
  }

  return hipSuccess;
}


// New function for 3D tensor copy blocks operation
extern "C" hipError_t
copy_blocks_3d(
    const void* src_data, void* dst_data, const int* h_src_block_ids, const int* h_dst_block_ids, int num_block_pairs,
    int prefix_dim, int src_blocks_dim, int dst_blocks_dim, int suffix_dim, int elem_size)
{
#ifdef USE_KERNEL
  // Allocate device memory for block IDs
  int* d_src_block_ids = NULL;
  int* d_dst_block_ids = NULL;

  CUDA_CHECK(cudaMalloc(&d_src_block_ids, num_block_pairs * sizeof(int)));
  CUDA_CHECK(cudaMalloc(&d_dst_block_ids, num_block_pairs * sizeof(int)));

  CUDA_CHECK(
      cudaMemcpyAsync(d_src_block_ids, h_src_block_ids, num_block_pairs * sizeof(int), cudaMemcpyHostToDevice, 0));
  CUDA_CHECK(
      cudaMemcpyAsync(d_dst_block_ids, h_dst_block_ids, num_block_pairs * sizeof(int), cudaMemcpyHostToDevice, 0));

  // Launch kernel with explicit strides
  cudaError_t result = copy_blocks_launcher_3d(
      src_data, dst_data, d_src_block_ids, d_dst_block_ids, num_block_pairs, prefix_dim, suffix_dim, elem_size,
      src_blocks_dim, dst_blocks_dim, 0);

  // Handle errors from kernel launch
  if (result != cudaSuccess) {
    cudaFree(d_src_block_ids);
    cudaFree(d_dst_block_ids);
    return result;
  }
#else
  hipError_t result = copy_blocks_memcpy_3d(
      src_data, dst_data, h_src_block_ids, h_dst_block_ids, num_block_pairs, prefix_dim, suffix_dim, elem_size,
      src_blocks_dim, dst_blocks_dim, 0);
#endif
  // Wait for completion
  CUDA_CHECK(hipStreamSynchronize(0));

#ifdef USE_KERNEL
  // Clean up
  cudaFree(d_src_block_ids);
  cudaFree(d_dst_block_ids);
#endif

  return hipSuccess;
}


// TODO: Refactor the driver code to take pointers for the device block_id arrays
// TODO: Maintain a blocking driver, but then also provide a non-blocking driver
//
// We will have N copies of the CopyStream struct which we will put in a reusable
// pool. Acquiring a CopyStream will let you perform a copy for a kv attention layer.
//
// From rust or python we'll execute this on a thread allowed to block. We'll await the
// cuda event for completion and report the return code on the driver.
//
// TODO: decide whether or not we need a pool of streams or use a single stream.
//
// We should be able to decouple this from the forward pass. The only condition is that
// a new forward pass can not start until the last copy has completed.
//
// To that end, we might want to tie this copy kernel to the stream used for the forward pass.
struct CopyStream {
  // Device block arrays
  int* d_src_blocks;
  int* d_dst_blocks;

  // Host copies of block arrays
  int* h_src_blocks;
  int* h_dst_blocks;

  int num_blocks;

  hipStream_t stream;
  hipEvent_t start_event;
  hipEvent_t stop_event;

  CopyStream(int num_layers, int num_blocks);
  ~CopyStream();

  void reset();
};

CopyStream::CopyStream(int num_layers, int num_blocks)
{
  hipError_t status;

  // Allocate device memory
  status = hipMalloc(&d_src_blocks, num_blocks * sizeof(int));
  if (status != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(status));
    return;
  }

  status = hipMalloc(&d_dst_blocks, num_blocks * sizeof(int));
  if (status != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(status));
    hipFree(d_src_blocks);
    return;
  }

  // Allocate host memory
  h_src_blocks = (int*)malloc(num_blocks * sizeof(int));
  h_dst_blocks = (int*)malloc(num_blocks * sizeof(int));
  if (!h_src_blocks || !h_dst_blocks) {
    fprintf(stderr, "Host memory allocation failed\n");
    if (h_src_blocks)
      free(h_src_blocks);
    hipFree(d_src_blocks);
    hipFree(d_dst_blocks);
    return;
  }

  status = hipStreamCreate(&stream);
  if (status != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(status));
    free(h_src_blocks);
    free(h_dst_blocks);
    hipFree(d_src_blocks);
    hipFree(d_dst_blocks);
    return;
  }

  // Create events
  status = hipEventCreateWithFlags(&start_event, hipEventDisableTiming);
  if (status != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(status));
    free(h_src_blocks);
    free(h_dst_blocks);
    hipFree(d_src_blocks);
    hipFree(d_dst_blocks);
  }

  status = hipEventCreateWithFlags(&stop_event, hipEventDisableTiming);
  if (status != hipSuccess) {
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(status));
    free(h_src_blocks);
    free(h_dst_blocks);
    hipFree(d_src_blocks);
    hipFree(d_dst_blocks);
  }
}

CopyStream::~CopyStream()
{
  free(h_src_blocks);
  free(h_dst_blocks);
  hipFree(d_src_blocks);
  hipFree(d_dst_blocks);
  hipEventDestroy(start_event);
  hipEventDestroy(stop_event);
}


extern "C" {
int cuda_malloc_host(void** ptr, size_t size);
int cuda_free_host(void* ptr);
int cuda_memcpy_async(void* dst, const void* src, size_t count, hipStream_t stream);

int
copy_stream_create(CopyStream** stream, int num_layers, int num_blocks)
{
  *stream = new CopyStream(num_layers, num_blocks);
  return 0;
}

int
copy_stream_destroy(CopyStream* stream)
{
  delete stream;
  return 0;
}


int
copy_stream_prepare_block_ids(CopyStream* cs, int* src_block_ids, int* dst_block_ids, int num_blocks)
{
  // Make host copies
  memcpy(cs->h_src_blocks, src_block_ids, num_blocks * sizeof(int));
  memcpy(cs->h_dst_blocks, dst_block_ids, num_blocks * sizeof(int));

  // Copy to device (for kernel-based implementation)
  CUDA_CHECK(
      hipMemcpyAsync(cs->d_src_blocks, src_block_ids, num_blocks * sizeof(int), hipMemcpyHostToDevice, cs->stream));
  CUDA_CHECK(
      hipMemcpyAsync(cs->d_dst_blocks, dst_block_ids, num_blocks * sizeof(int), hipMemcpyHostToDevice, cs->stream));

  cs->num_blocks = num_blocks;

  return 0;
}

int
copy_stream_launch(
    CopyStream* cs, const void* src_data, void* dst_data, int prefix_dim, int suffix_dim, int elem_size,
    int src_block_dim, int dst_block_dim)
{
  return copy_blocks_launcher_3d(
      src_data, dst_data, cs->d_src_blocks, cs->d_dst_blocks, cs->num_blocks, prefix_dim, suffix_dim, elem_size,
      src_block_dim, dst_block_dim, cs->stream);
}

int
copy_stream_memcpy(
    CopyStream* cs, const void* src_data, void* dst_data, int prefix_dim, int suffix_dim, int elem_size,
    int src_block_dim, int dst_block_dim)
{
  return copy_blocks_memcpy_3d(
      src_data, dst_data, cs->h_src_blocks, cs->h_dst_blocks, cs->num_blocks, prefix_dim, suffix_dim, elem_size,
      src_block_dim, dst_block_dim, cs->stream);
}

int
copy_stream_sync(CopyStream* cs)
{
  // sync on the event
  CUDA_CHECK(hipStreamSynchronize(cs->stream));
  return hipSuccess;
}

int
cuda_malloc_host(void** ptr, size_t size)
{
  CUDA_CHECK(hipHostAlloc(ptr, size, hipHostMallocDefault));
  return hipSuccess;
}

int
cuda_free_host(void* ptr)
{
  CUDA_CHECK(hipHostFree(ptr));
  return hipSuccess;
}

int
cuda_memcpy_async(void* dst, const void* src, size_t count, hipStream_t stream)
{
  CUDA_CHECK(hipMemcpyAsync(dst, src, count, hipMemcpyDefault, stream));
  return hipSuccess;
}

int
cuda_memcpy_sync(void* dst, const void* src, size_t count)
{
  CUDA_CHECK(hipMemcpy(dst, src, count, hipMemcpyDefault));
  return hipSuccess;
}
}

/// This accepts a 6D tensor with dimensions that represent a tensor to be distributed
/// across tensor parallel ranks.
///
/// The dimensions of the source tensor are expected to be:
/// dims[0]: kv or block (depending on KvLayout)
/// dims[1]: block or kv (depending on KvLayout)
/// dims[2]: block_size (sequence length) # aka bs
/// dims[3]: scatter_factor (dst_tp_size / src_tp_size)
/// dims[4]: num_heads / (src_tp_size * scatter_factor) # aka dst_num_heads or dnh
/// dims[5]: head_size # aka hs
///
/// The permutation applied is (3, 0, 1, 2, 4, 5) which transforms
/// the tensor:
///  - from: [kv/block, block/kv, bs, scatter_factor, dnh, hs] to
///  - to:   [scatter_factor, kv/block, block/kv, bs, dnh, hs].
///
/// This transformation effectively distributes the heads dimension across
/// tensor parallel ranks, where we transform from src_tp_size to dst_tp_size,
/// with dst_tp_size > src_tp_size.
int
permute_scatter_memcpy(
    const void* src,           // source data
    void* dst,                 // destination data
    const uint32_t* dims,      // 6d dimensions of source tensor
    uint32_t num_dims,         // semi-redundant, size of the dims array, must be 6
    uint32_t elem_size,        // element size in bytes
    uint32_t block_dim_index,  // which dimension represents blocks
    uint32_t src_block_dim,    // the dimension of the source blocks
    uint32_t dst_block_dim,    // the dimension of the destination blocks
    int* src_block_ids,        // from state: the block IDs to copy
    int* dst_block_ids,        // from state: the block IDs to copy
    uint32_t num_blocks,       // from state: the number of blocks to copy
    hipStream_t stream        // from state: the stream to use
)
{
  if (num_dims != 6) {
    printf("ERROR: num_dims must be 6\n");
    return -1;
  }

  if (block_dim_index != 0 && block_dim_index != 1) {
    printf("ERROR: block_dim_index must be 0 or 1\n");
    return -2;
  }

  uint32_t kv_dim_index = block_dim_index == 0 ? 1 : 0;

  // expect dims[block_dim_index] == src_block_dim
  // expect dims[kv_dim_index] == 2
  if (dims[block_dim_index] != src_block_dim) {
    printf("ERROR: dims[block_dim_index] must be equal to src_block_dim\n");
    return -3;
  }

  if (dims[kv_dim_index] != 2) {
    printf("ERROR: dims[kv_dim_index] must be 2\n");
    return -4;
  }

  size_t src_shape[5];
  size_t dst_shape[5];

  src_shape[block_dim_index] = src_block_dim;
  src_shape[kv_dim_index] = dims[kv_dim_index];
  src_shape[2] = dims[2];
  src_shape[3] = dims[3];
  src_shape[4] = dims[4] * dims[5];

  dst_shape[0] = dims[3];  // scatter factor
  dst_shape[block_dim_index + 1] = dst_block_dim;
  dst_shape[kv_dim_index + 1] = dims[kv_dim_index];
  dst_shape[3] = dims[2];  // block size
  dst_shape[4] = dims[4] * dims[5];

  size_t src_strides[5];
  size_t dst_strides[5];

  src_strides[4] = elem_size;
  dst_strides[4] = elem_size;

  // Compute source strides recursively (row-major order)
  for (int i = 3; i >= 0; i--) {
    src_strides[i] = src_strides[i + 1] * src_shape[i + 1];
  }

  // Compute destination strides based on permuted dimensions
  for (int i = 3; i >= 0; i--) {
    dst_strides[i] = dst_strides[i + 1] * dst_shape[i + 1];
  }

#ifdef DEBUG
  printf("src_shape: ");
  for (int i = 0; i < 5; i++) {
    printf("%zu ", src_shape[i]);
  }
  printf("\n");

  printf("src_strides: ");
  for (int i = 0; i < 5; i++) {
    printf("%zu ", src_strides[i]);
  }
  printf("\n");

  printf("dst_shape: ");
  for (int i = 0; i < 5; i++) {
    printf("%zu ", dst_shape[i]);
  }
  printf("\n");

  printf("dst_strides: ");
  for (int i = 0; i < 5; i++) {
    printf("%zu ", dst_strides[i]);
  }
  printf("\n");
#endif

  size_t copy_size_bytes = dims[4] * dims[5] * elem_size;

  // we will start by computing the full offsets for each inner copy blocks
  size_t src_idx[5];
  size_t dst_idx[5];

  // notes:
  // - in the outer two loops, the index for the dst is shifted by one since we moved the
  //   scatter dimension to the front [0]

  const char* src_ptr = (const char*)src;
  char* dst_ptr = (char*)dst;

  // loop over blocks
  for (int block = 0; block < num_blocks; block++) {
    src_idx[block_dim_index] = block;
    dst_idx[block_dim_index + 1] = block;
    // loop over the kv dimension
    for (int kv = 0; kv < src_shape[kv_dim_index]; kv++) {
      src_idx[kv_dim_index] = kv;
      dst_idx[kv_dim_index + 1] = kv;
      // loop over block size
      for (int block_size = 0; block_size < src_shape[2]; block_size++) {
        src_idx[2] = block_size;
        dst_idx[3] = block_size;
        // loop over scatter factor
        for (int scatter = 0; scatter < src_shape[3]; scatter++) {
          src_idx[3] = scatter;
          dst_idx[0] = scatter;

          src_idx[4] = 0;
          dst_idx[4] = 0;

          size_t src_offset = 0;
          size_t dst_offset = 0;

          for (int i = 0; i < 5; i++) {
            src_offset += src_idx[i] * src_strides[i];
            dst_offset += dst_idx[i] * dst_strides[i];
          }

          auto rc =
              hipMemcpyAsync(dst_ptr + dst_offset, src_ptr + src_offset, copy_size_bytes, hipMemcpyDefault, stream);

          if (rc != hipSuccess) {
            printf("ERROR: hipMemcpyAsync failed with error code %d\n", rc);
            return -5;
          }
        }
      }
    }
  }

  return 0;
}

// Updated C API wrapper for the permutation function
extern "C" int
copy_stream_scatter(
    CopyStream* cs,            // the copy stream
    const void* src_data,      // the source data (single layer)
    void* dst_data,            // the destination data (single layer)
    const uint32_t* dims,      // 6d dimensions of source tensor
    uint32_t num_dims,         // semi-redundant, size of the dims array, must be 6
    uint32_t elem_size,        // element size in bytes
    uint32_t block_dim_index,  // which dimension represents blocks; either 0 or 1
    uint32_t src_block_dim,    // number of blocks in the src tensor (should match dims[block_dim_index])
    uint32_t dst_block_dim     // number of blocks in the dst tensor
)
{
  return permute_scatter_memcpy(
      src_data,          //
      dst_data,          //
      dims,              //
      num_dims,          //
      elem_size,         //
      block_dim_index,   //
      src_block_dim,     //
      dst_block_dim,     //
      cs->h_src_blocks,  //
      cs->h_dst_blocks,  //
      cs->num_blocks,    //
      cs->stream         //
  );
}
